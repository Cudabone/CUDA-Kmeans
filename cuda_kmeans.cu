
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

void cudaErrorCheck(hipError_t err, const char *s);
__device__ inline float euclidean_dist(float *d_Clusters, 
										float *d_Objects, 
										int numClusters,
										int numObjects,
										int numCoords,
										int objectNum,
										int clusterNum)
{
	float dist = 0.0;

	int i;
	int objIdx,clusterIdx;
	for(i = 0; i < numCoords; i++)
	{
		//Index for object and cluster
		objIdx = numObjects*i + objectNum;
		clusterIdx = numClusters*i + clusterNum;

		dist += (d_Objects[objIdx] - d_Clusters[clusterIdx]) * 
			(d_Objects[objIdx]- d_Clusters[clusterIdx]);
	}
	return dist;
}
__global__ void find_nearest_cluster(float *d_Clusters,
										float *d_Objects,
										float *d_Deltas,
										float *d_Membership,
										int numClusters,
										int numObjects,
										int numCoords)
{
	//Shared to store delta's for block
	extern __shared__ float delta[]; // 
	//Thread index into objects
	unsigned int objectNum = blockDim.x * blockIdx.x + threadIdx.x;

	//Ensure that thread within # objects and block has a cluster
	if(objectNum >= numObjects)
		return;

	delta[threadIdx.x] = 0.0;
	__syncthreads();

	//find distance
    int   index, i;
    float dist, min_dist;

    /* find the cluster id that has min distance to object */
    index    = 0;
    min_dist =
		euclidean_dist(d_Clusters,d_Objects,numClusters,numObjects,numCoords,objectNum,0);

    for (i=1; i<numClusters; i++) {
        dist = 
		euclidean_dist(d_Clusters,d_Objects,numClusters,numObjects,numCoords,objectNum,i);
        /* no need square root */
        if (dist < min_dist) { /* find the min and its array index */
            min_dist = dist;
            index    = i;
        }
    }
	//Each thread now knows the closest cluster to its object
	if(d_Membership[objectNum] != index)
		delta[objectNum] = 1.0;

	//TODO Can use shared mem for this if better
	//Assign Membership for Object
	d_Membership[objectNum] = index;

	//Reduce Deltas for block
	__syncthreads();
	for(unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
	{
		if(threadIdx.x < stride)
			delta[threadIdx.x] += delta[threadIdx.x + stride];  
		__syncthreads();
	}

	//Assign delta for block in device deltas
	if(threadIdx.x == 0)
		d_Deltas[blockIdx.x] = delta[0];
}
/*----< seq_kmeans() >-------------------------------------------------------*/
/* return an array of cluster centers of size [numClusters][numCoords]       */
int cuda_kmeans(float **objects,      /* in: [numObjs][numCoords] */
               int     numCoords,    /* no. features */
               int     numObjs,      /* no. objects */
               int     numClusters,  /* no. clusters */
               float   threshold,    /* % objects change membership */
               int    *membership,   /* out: [numObjs] */
               float **clusters)     /* out: [numClusters][numCoords] */
{
	float *d_Clusters;
	float *d_Objects;
	float *h_Deltas;
	float *d_Deltas;
	float *d_Membership;
	float **newClusters;
	int *newClusterSize;

	int blocksize = 32;
	int numblocks = ceil(numObjs/blocksize);

	h_Deltas = (float *)malloc(blocksize*sizeof(float));
	//Allocate and Initialize newClusterSize to 0's
	newClusterSize = (int *)calloc(numClusters,sizeof(int));

	//Create new clusters as is done in original algorithm
    newClusters    = (float**) malloc(numClusters *            sizeof(float*));
    assert(newClusters != NULL);
    newClusters[0] = (float*)  calloc(numClusters * numCoords, sizeof(float));
    assert(newClusters[0] != NULL);

	int i;
	//Initialize all pointers for newClusters
    for (i=1; i<numClusters; i++)
        newClusters[i] = newClusters[i-1] + numCoords;

	cudaErrorCheck(hipMalloc((void
					**)&d_Clusters,numClusters*numCoords*sizeof(float)),
				"CMalloc d_Clusters");
	cudaErrorCheck(hipMalloc((void
					**)&d_Objects,numObjs*numCoords*sizeof(float)),
			"Cmalloc d_Objects");
	cudaErrorCheck(hipMalloc((void
					**)&d_Deltas,blocksize*sizeof(float)),
			"Cmalloc d_Deltas");
	cudaErrorCheck(hipMalloc((void
					**)&d_Membership,numObjs*sizeof(float)),
			"Cmalloc d_Membership");
	//cudaErrorCheck(cudaMemcpy2DToArray(,cudaMemcpyHostToDevice))
	int loop = 0;
	int j;
	int index;
	float delta;

	// Initialize Membership
	for(i = 0; i < numObjs; i++)
		membership[i] = -1;

	cudaErrorCheck(hipMemcpy((void *)d_Membership,(const void
					*)membership,numObjs*sizeof(float),hipMemcpyHostToDevice),"Members to device");

	do{
		delta = 0.0;
		//TODO Can pretty much implement this entire loop i think in CUDA only
		//find_nearest_cluster
		hipDeviceSynchronize();
		/* find the array index of nestest cluster center */
		//TODO Create and Replace find_nearest cluster for Cuda

		//index = find_nearest_cluster(numClusters, numCoords, objects[i],clusters);
		find_nearest_cluster<<<numblocks,blocksize,blocksize>>>
			(d_Clusters,d_Objects,d_Deltas,d_Membership,numClusters,numObjs,numCoords);
		hipDeviceSynchronize();
		cudaErrorCheck(hipMemcpy((void *)h_Deltas,(const void
						*)d_Deltas,blocksize*sizeof(float),hipMemcpyDeviceToHost),"deltas to host");
		//Sum all deltas from each block
		for(i = 0; i < blocksize; i++)
			delta += h_Deltas[i];

		//The Rest is mostly left unchanged

		//TODO Look at rest
		/* if membership changes, increase delta by 1 */
		if (membership[i] != index) delta += 1.0;

		/* assign the membership to object i */
		membership[i] = index;

		/* update new cluster center : sum of objects located within */
		for(i = 0; i < numObjs; i++)
		{
			newClusterSize[index]++;
			for (j=0; j<numCoords; j++)
				newClusters[index][j] += objects[i][j];
		}
		/* average the sum and replace old cluster center with newClusters */
		for (i=0; i<numClusters; i++) {
			for (j=0; j<numCoords; j++) {
				if (newClusterSize[i] > 0)
					clusters[i][j] = newClusters[i][j] / newClusterSize[i];
				newClusters[i][j] = 0.0;   /* set back to 0 */
			}
			newClusterSize[i] = 0;   /* set back to 0 */
		}

		delta /= numObjs;
	} while (delta > threshold && loop++ < 500);

	//Free All Cuda and C Pointers

	hipFree(d_Clusters);	
	hipFree(d_Objects);
	hipFree(d_Deltas);
	hipFree(d_Membership);

	free(h_Deltas);
    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

	return 1;
}
void cudaErrorCheck(hipError_t err, const char *s)
{
	if(err != hipSuccess)
	{
		printf("%s error: %s\n",s,hipGetErrorString(err));
		exit(0);
	}
}
